#include<iostream>
#include <hip/hip_runtime.h>
#include<stdio.h>

using namespace std;


#define LINE 700
#define TYPE int


__global__ void two_matrix_times(const TYPE* a, const TYPE* b, TYPE* c,int rows, int cols){
    // 此代码用于求两个矩阵之间的二范数距离
    int x = blockIdx.x*blockDim.x + threadIdx.x;//x,y 在全局kernal中的位置
    if(x < rows*cols){
        // (*c) += (a[x] - b[x])*(a[x] - b[x]); //这种方式会产生竞争
        atomicAdd(c, (a[x] - b[x])*(a[x] - b[x]));
        __syncthreads();//等待所有的线程计算完成
    }
    
}

void print_matrixs(const TYPE* res, int rows, int cols){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            cout<<" "<<res[rows*i + j];
        }
        cout<<endl;
    }
    
}

void init_matrix(TYPE* data, int rows, int cols, TYPE num){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            data[i*rows+j] = num;
        }
    }
}

int main(int argc, char const *argv[])
{

    hipEvent_t start, stop; //创建event
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t size = LINE*LINE*sizeof(TYPE);
    TYPE *D_a, *D_b, *D_c;
    TYPE *H_a, *H_b, *H_c;


    H_a = (TYPE *)malloc(size); // 在Host中分配内存，否则会报错
    H_b = (TYPE *)malloc(size); // 在Host中分配内存，否则会报错
    H_c = (TYPE *)malloc(sizeof(TYPE)); // 在Host中分配内存，否则会报错
    *H_c = 0;
    // cout<<"before malloc "<<*H_c<<endl;
    hipMalloc((void **)&D_a, size);  //在GPU上分配内存
    hipMalloc((void **)&D_b, size);
    hipMalloc((void **)&D_c, sizeof(TYPE));

    init_matrix(H_a, LINE, LINE,3);//初始化矩阵
    init_matrix(H_b, LINE, LINE,1);//初始化矩阵
    
    hipMemcpy(D_a, H_a, size, hipMemcpyHostToDevice); //将host上的内容copy到device上。注意这个函数的位置
    hipMemcpy(D_b, H_b, size, hipMemcpyHostToDevice);
    hipMemcpy(D_c, H_c, sizeof(TYPE), hipMemcpyHostToDevice);

    hipEventRecord(start, 0); //开始计时
    two_matrix_times<<<958, 512>>>(D_a,D_b,D_c,LINE, LINE); //执行kernal
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop); //结束计时，且将运行时间保存在elapsedTime
    cout<<"GPU cost time: "<<elapsedTime/1000.0<<" S"<<endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(H_c, D_c, sizeof(TYPE),hipMemcpyDeviceToHost);  // 将运算结果返回给Host
    cout<<"Final result is "<<*H_c<<endl;

    hipFree(D_a);
    hipFree(D_b);
    hipFree(D_c);

    return 0;
}
